#include "helper.h"

const std::string outfolderopencv="./images/output/opencv/";
const std::string outfoldercudanpp="./images/output/cudanpp/";

timeval tiStart;
void twachstart()
{
    gettimeofday(&tiStart, 0);
}

int twatchend() {
    timeval tiEnd;
    gettimeofday(&tiEnd, 0);
    int t = (tiEnd.tv_sec - tiStart.tv_sec) * 1000000 + tiEnd.tv_usec - tiStart.tv_usec;
    tiStart = tiEnd;
    return t;
}

std::vector<std::string> srcImagesFilenames()
{   
    const char* imgFolder ="./images/src/";        
    std::vector<std::string> filenames;
    DIR *dr;
    struct dirent *en;
    dr = opendir(imgFolder); 
    if (dr) 
    {
        while ((en = readdir(dr)) != NULL) 
        {            
            std::string fullname(imgFolder);
            if (strstr(en->d_name,".JPG") || strstr(en->d_name,".jpg") || strstr(en->d_name,".PNG") || strstr(en->d_name,".png")) 
            {
                fullname+=en->d_name;
                std::string fname(fullname); //assign a value from *char on a constructor                   
                filenames.push_back(fname);
            }
        }
        closedir(dr);
    }    
    else if (dr == NULL) 
    {        
        perror("An error occurred while trying to access the specified folder.");       
        std::cout << "Specidied Folder may not exist:"<< imgFolder << "\n";       
    }
    return filenames;
}


int opencvGrayscale(std::string flocation)
{    
    cv::Mat srcimg = cv::imread(flocation);
    cv::Mat grayimg;

    twachstart();
    cv::cvtColor(srcimg, grayimg, cv::COLOR_BGR2GRAY);
    
    #ifdef _WIN32 
       std::string fname = flocation.substr(flocation.find_last_of("/\\") + 1);   
    #else
       std::string fname = flocation.substr(flocation.find_last_of("/") + 1);  
    #endif   
    int timeeaplsed=twatchend();     
    cv::imwrite(outfolderopencv+fname,grayimg);
    std::cout << "Image size in bytes "<<fname <<" Source:"<< srcimg.step[0] * srcimg.rows<<" OpenCV Gray:" << grayimg.step[0] * grayimg.rows << " Time elapsed:" << timeeaplsed;

    return timeeaplsed;
}

int convertToGrayCuda(std::string flocation)
{    
    cv::Mat srcimg = cv::imread(flocation, cv::IMREAD_COLOR);

    twachstart();

    Npp8u* srcimgdev;
    hipMalloc((void**)&srcimgdev, srcimg.total() * srcimg.elemSize());
    Npp8u* outimgdev;
    hipMalloc((void**)&outimgdev, srcimg.total());

    //Copy from the host memory input image to the CUDA device memory object
    hipMemcpy(srcimgdev, srcimg.data, srcimg.total() * srcimg.elemSize(), hipMemcpyHostToDevice);
    
    NppiSize sizeROI;
    sizeROI.width = srcimg.cols;
    sizeROI.height = srcimg.rows;
    //set conversion coefficients 
    const Npp32f convCoeff[3]={0.299,0.587,0.114};   
    //Perform the grayscale conversion
    nppiColorToGray_8u_C3C1R(srcimgdev, srcimg.step, outimgdev, srcimg.cols, sizeROI, convCoeff);

    // Allocate memory host memory output image
    cv::Mat outimghost(srcimg.rows, srcimg.cols, CV_8UC1);

    // Copy the result from the CUDA device memory to the host memory
    hipMemcpy(outimghost.data, outimgdev, srcimg.total(), hipMemcpyDeviceToHost);
            
    // Release the memory
    hipFree(srcimgdev);
    hipFree(outimgdev);

    int timeeaplsed=twatchend(); 
    
    // Save the grayscale image
    #ifdef _WIN32 
       std::string fname = flocation.substr(flocation.find_last_of("/\\") + 1);   
    #else
       std::string fname = flocation.substr(flocation.find_last_of("/") + 1);  
    #endif

    cv::imwrite(outfoldercudanpp+fname,outimghost);    
    std::cout <<" CUDA Gray:" << outimghost.step[0] * outimghost.rows << " Time elapsed:" << timeeaplsed;
    
    return timeeaplsed;
}

bool printNPPinfo(int argc, char *argv[])
{
  const NppLibraryVersion *libVer = nppGetLibVersion();

  printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor,libVer->build);

  int driverVersion, runtimeVersion;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  printf("  CUDA Driver  Version: %d.%d\n", driverVersion / 1000,(driverVersion % 100) / 10);
  printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

  // Min spec is SM 1.0 devices
  bool bVal = checkCudaCapabilities(1, 0);
  return bVal;
}

int main(int argc, char *argv[])
{
  findCudaDevice(argc, (const char **)argv);
  
  if (printNPPinfo(argc, argv) == false)
  {
      exit(EXIT_SUCCESS);
  }

  long ttotalOpencv=0,ttotalCudanpp=0;  
  auto fnames = srcImagesFilenames(); 
  std::cout.imbue(std::locale(""));
  for(int i=0;i<fnames.size();i++)
  {         
     std::string fn(fnames[i]); 

     ttotalOpencv+=opencvGrayscale(fn);         
     ttotalCudanpp+=convertToGrayCuda(fn);     
     std::cout << "\n";     
  }
  std::cout <<"---------------------------------------------------------\n";
  std::cout<< "Total elapsed time with OPNCV library: " << ttotalOpencv<<"\n";
  std::cout<< "Total elapsed time with CUDA NPP library: " << ttotalCudanpp<<"\n";

  return 0;
}
